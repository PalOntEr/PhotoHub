#include "hip/hip_runtime.h"
﻿// kernel.cu
//Compiler: nvcc
//Bash Line: nvcc -o kernel.dll --shared kernel.cu
//BGR image format
//pixel_index + 2 red
//pixel_index + 1 green
//pixel_index blue

#include <hip/hip_math_constants.h>
#include <hiprand/hiprand_kernel.h>

extern "C" __global__ void generate_histogram_kernel(int width, int height, unsigned char* pixels, int* red_histogram, int* green_histogram, int* blue_histogram) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width * height) {
        int pixel_index = idx * 3;
        atomicAdd(&red_histogram[pixels[pixel_index + 2]], 1);
        atomicAdd(&green_histogram[pixels[pixel_index + 1]], 1);
        atomicAdd(&blue_histogram[pixels[pixel_index]], 1);
    }
}

extern "C" __global__ void grayscale_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        unsigned char grayscale_value = (pixels[pixel_index] + pixels[pixel_index + 1] + pixels[pixel_index + 2]) / 3;
        result[pixel_index] = grayscale_value;
        result[pixel_index + 1] = grayscale_value;
        result[pixel_index + 2] = grayscale_value;
    }
}
extern "C" __global__ void negative_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        result[pixel_index] = 255 - pixels[pixel_index];
        result[pixel_index + 1] = 255 - pixels[pixel_index + 1];
        result[pixel_index + 2] = 255 - pixels[pixel_index + 2];
    }
}
extern "C" __global__ void sepia_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        unsigned char gray_value = static_cast<unsigned char>(pixels[pixel_index] * 0.3f + pixels[pixel_index + 1] * 0.59f + pixels[pixel_index + 2] * 0.11f);
        result[pixel_index + 2] = min(255, gray_value + 40);
        result[pixel_index + 1] = min(255, gray_value + 20);
        result[pixel_index] = min(255, gray_value);
    }
}
extern "C" __global__ void gaussian_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        float pr = 0.0f;
        float pg = 0.0f;
        float pb = 0.0f;
        float total_weight = 0.0f;

        int kernel_radius = 1;
        float kernel[3][3] = {
            {1.0f / 16, 1.0f / 8, 1.0f / 16},
            {1.0f / 8, 1.0f / 4, 1.0f / 8},
            {1.0f / 16, 1.0f / 8, 1.0f / 16}
        };

        for (int ky = -kernel_radius; ky <= kernel_radius; ky++) {
            for (int kx = -kernel_radius; kx <= kernel_radius; kx++) {
                int neighbor_x = (idx % width) + kx;
                int neighbor_y = (idx / width) + ky;
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    int neighbor_index = (neighbor_y * width + neighbor_x) * 3;
                    float weight = kernel[ky + kernel_radius][kx + kernel_radius];
                    pr += pixels[neighbor_index + 2] * weight;
                    pg += pixels[neighbor_index + 1] * weight;
                    pb += pixels[neighbor_index] * weight;
                    total_weight += weight;
                }
            }
        }

        result[pixel_index + 2] = static_cast<unsigned char>(pr / total_weight);
        result[pixel_index + 1] = static_cast<unsigned char>(pg / total_weight);
        result[pixel_index] = static_cast<unsigned char>(pb / total_weight);
    }
}
extern "C" __global__ void emboss_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        float pr = 0.0f;
        float pg = 0.0f;
        float pb = 0.0f;

        int kernel_radius = 1;
        float mask1[3][3] = {
            {0, 1, 0},
            {0, 0, 0},
            {0, -1, 0}
        };
        float mask2[3][3] = {
            {1, 0, 0},
            { 0, 0, 0 },
            {0, 0, -1}
        };
        float mask3[3][3] = {
            {0, 0, 0},
            {1, 0, -1},
            {0, 0, 0}
        };
        float mask4[3][3] = {
            {0, 0, 1},
            {0, 0, 0},
            {-1, 0, 0}
        };

        for (int ky = -kernel_radius; ky <= kernel_radius; ky++) {
            for (int kx = -kernel_radius; kx <= kernel_radius; kx++) {
                int neighbor_x = (idx % width) + kx;
                int neighbor_y = (idx / width) + ky;
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    int neighbor_index = (neighbor_y * width + neighbor_x) * 3;
                    float weight = mask1[ky + kernel_radius][kx + kernel_radius];
					weight += mask2[ky + kernel_radius][kx + kernel_radius];
					weight += mask3[ky + kernel_radius][kx + kernel_radius];
					weight += mask4[ky + kernel_radius][kx + kernel_radius];
                    pr += pixels[neighbor_index + 2] * weight;
                    pg += pixels[neighbor_index + 1] * weight;
                    pb += pixels[neighbor_index] * weight;
                }
            }
        }

        result[pixel_index + 2] = min(max(static_cast<unsigned char>(pr) + 128, 0), 255);
        result[pixel_index + 1] = min(max(static_cast<unsigned char>(pg) + 128, 0), 255);
        result[pixel_index] = min(max(static_cast<unsigned char>(pb) + 128, 0), 255);
    }

}
extern "C" __global__ void edge_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int kernel_radius = 1;
        float sobel_x[3][3] = {
            {1, 2, 1},
            {0, 0, 0},
            {-1, -2, -1}
        };
        float sobel_y[3][3] = {
            {-1, 0, 1},
            {-2, 0, 2},
            {-1, 0, 1}
        };

        int pixel_index = idx * 3;
        int Lx = 0;
        int Ly = 0;

        for (int ky = -kernel_radius; ky <= kernel_radius; ky++) {
            for (int kx = -kernel_radius; kx <= kernel_radius; kx++) {
                int neighbor_x = (idx % width) + kx;
                int neighbor_y = (idx / width) + ky;
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    int neighbor_index = (neighbor_y * width + neighbor_x) * 3;

                    unsigned char grayscale_value = (pixels[neighbor_index] + pixels[neighbor_index + 1] + pixels[neighbor_index + 2]) / 3;

                    Lx += sobel_x[ky + kernel_radius][kx + kernel_radius] * grayscale_value;
                    Ly += sobel_y[ky + kernel_radius][kx + kernel_radius] * grayscale_value;
                }
            }
        }

        float gradient_magnitude = sqrtf(Lx * Lx + Ly * Ly);

        gradient_magnitude = gradient_magnitude > 50 ? 255 : 0;

        result[pixel_index + 2] = static_cast<unsigned char>(min(max(gradient_magnitude, 0.0f), 255.0f));
        result[pixel_index + 1] = static_cast<unsigned char>(min(max(gradient_magnitude, 0.0f), 255.0f));
        result[pixel_index] = static_cast<unsigned char>(min(max(gradient_magnitude, 0.0f), 255.0f));
    }
}
extern "C" __global__ void thermal_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {  
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
  
    if (idx < width * height) {  
        int pixel_index = idx * 3;  
        unsigned char gray_value = (pixels[pixel_index] + pixels[pixel_index + 1] + pixels[pixel_index + 2]) / 3;  
        unsigned char r, g, b;  
  
        if (gray_value < 64) {  
            r = 4 * gray_value;  
            g = 0;  
            b = 0;  
        } else if (gray_value < 128) {  
            r = 255 - 4 * (gray_value - 64);  
            g = 4 * (gray_value - 64);  
            b = 0;  
        } else if (gray_value < 192) {  
            r = 0;  
            g = 255 - 4 * (gray_value - 128);  
            b = 4 * (gray_value - 128);  
        } else {  
            r = 0;  
            g = 4 * (gray_value - 192);  
            b = 255;  
        }  
  
        result[pixel_index + 2] = r;  
        result[pixel_index + 1] = g;  
        result[pixel_index] = b;  
    }  
}
extern "C" __global__ void contrast_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;

		float contrast = 2.0f;

		result[pixel_index + 2] = min(max(static_cast<unsigned char>(contrast * (pixels[pixel_index + 2] - 128) + 128), 0), 255);
		result[pixel_index + 1] = min(max(static_cast<unsigned char>(contrast * (pixels[pixel_index + 1] - 128) + 128), 0), 255);
		result[pixel_index] = min(max(static_cast<unsigned char>(contrast * (pixels[pixel_index] - 128) + 128), 0), 255);
    }
}
extern "C" __global__ void sharpen_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        float pr = 0.0f;
        float pg = 0.0f;
        float pb = 0.0f;
        float total_weight = 0.0f;

        int kernel_radius = 1;
        float kernel[3][3] = {
            {1.0f / 16, 1.0f / 8, 1.0f / 16},
            {1.0f / 8, 1.0f / 4, 1.0f / 8},
            {1.0f / 16, 1.0f / 8, 1.0f / 16}
        };

        for (int ky = -kernel_radius; ky <= kernel_radius; ky++) {
            for (int kx = -kernel_radius; kx <= kernel_radius; kx++) {
                int neighbor_x = (idx % width) + kx;
                int neighbor_y = (idx / width) + ky;
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    int neighbor_index = (neighbor_y * width + neighbor_x) * 3;
                    float weight = kernel[ky + kernel_radius][kx + kernel_radius];
                    pr += pixels[neighbor_index + 2] * weight;
                    pg += pixels[neighbor_index + 1] * weight;
                    pb += pixels[neighbor_index] * weight;
                    total_weight += weight;
                }
            }
        }

		int coarse[3] = { 0 };

        coarse[0] = static_cast<unsigned char>(pr / total_weight);
		coarse[1] = static_cast<unsigned char>(pg / total_weight);
		coarse[2] = static_cast<unsigned char>(pb / total_weight);

		result[pixel_index + 2] = static_cast<unsigned char>((pixels[pixel_index + 2] * 1.5) - (coarse[0] * 0.5));
		result[pixel_index + 1] = static_cast<unsigned char>((pixels[pixel_index + 1] * 1.5) - (coarse[1] * 0.5));
		result[pixel_index] = static_cast<unsigned char>((pixels[pixel_index] * 1.5) - (coarse[2] * 0.5));

    }
}
extern "C" __global__ void noise_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
		int noise = 20;
		unsigned int seed = 0;

        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        result[pixel_index + 2] = min(max(static_cast<unsigned char>(pixels[pixel_index + 2] + ((hiprand(&state) % (2 * noise)) - noise) / 3), 0), 255);
        result[pixel_index + 1] = min(max(static_cast<unsigned char>(pixels[pixel_index + 1] + ((hiprand(&state) % (2 * noise)) - noise) / 3), 0), 255);
        result[pixel_index] = min(max(static_cast<unsigned char>(pixels[pixel_index] + ((hiprand(&state) % (2 * noise)) - noise) / 3), 0), 255);
    }
}
extern "C" __global__ void tilt_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int pixel_index = idx * 3;
        int x = idx % width;
        int y = idx / width;

        float center_x = width / 2.0f;
        float center_y = height / 2.0f;
		float distance = sqrtf((x - center_x) * (x - center_x) + (y - center_y) * (y - center_y));

		float blur_radius = min(height / 3, width / 3);

        float blur_amount = distance <= blur_radius ? 0 : distance;

        if (blur_amount > 0.0f) {
            float pr = 0.0f, pg = 0.0f, pb = 0.0f;
            float total_weight = 0.0f;
            int kernel_radius = 1;
            float kernel[3][3] = {
                {1.0f / 16, 1.0f / 8, 1.0f / 16},
                {1.0f / 8, 1.0f / 4, 1.0f / 8},
                {1.0f / 16, 1.0f / 8, 1.0f / 16}
            };

            for (int ky = -kernel_radius; ky <= kernel_radius; ky++) {
                for (int kx = -kernel_radius; kx <= kernel_radius; kx++) {
                    int neighbor_x = x + kx;
                    int neighbor_y = y + ky;
                    if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                        int neighbor_index = (neighbor_y * width + neighbor_x) * 3;
                        float weight = kernel[ky + kernel_radius][kx + kernel_radius];
                        pr += pixels[neighbor_index + 2] * weight;
                        pg += pixels[neighbor_index + 1] * weight;
                        pb += pixels[neighbor_index] * weight;
                        total_weight += weight;
                    }
                }
            }

            result[pixel_index + 2] = static_cast<unsigned char>(pr / total_weight);
            result[pixel_index + 1] = static_cast<unsigned char>(pg / total_weight);
            result[pixel_index] = static_cast<unsigned char>(pb / total_weight);
        }
        else {
            result[pixel_index + 2] = pixels[pixel_index + 2];
            result[pixel_index + 1] = pixels[pixel_index + 1];
            result[pixel_index] = pixels[pixel_index];
        }
    }
}
extern "C" __global__ void sketch_filter_kernel(int width, int height, unsigned char* pixels, unsigned char* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
        int kernel_radius = 1;
        float sobel_x[3][3] = {
            {1, 2, 1},
            {0, 0, 0},
            {-1, -2, -1}
        };
        float sobel_y[3][3] = {
            {-1, 0, 1},
            {-2, 0, 2},
            {-1, 0, 1}
        };

        int pixel_index = idx * 3;
        int Lx = 0;
        int Ly = 0;

        for (int ky = -kernel_radius; ky <= kernel_radius; ky++) {
            for (int kx = -kernel_radius; kx <= kernel_radius; kx++) {
                int neighbor_x = (idx % width) + kx;
                int neighbor_y = (idx / width) + ky;
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    int neighbor_index = (neighbor_y * width + neighbor_x) * 3;

                    unsigned char grayscale_value = (pixels[neighbor_index] + pixels[neighbor_index + 1] + pixels[neighbor_index + 2]) / 3;

                    Lx += sobel_x[ky + kernel_radius][kx + kernel_radius] * grayscale_value;
                    Ly += sobel_y[ky + kernel_radius][kx + kernel_radius] * grayscale_value;
                }
            }
        }

        float gradient_magnitude = sqrtf(Lx * Lx + Ly * Ly);

        result[pixel_index + 2] = static_cast<unsigned char>(min(max(gradient_magnitude, 0.0f), 255.0f));
        result[pixel_index + 1] = static_cast<unsigned char>(min(max(gradient_magnitude, 0.0f), 255.0f));
        result[pixel_index] = static_cast<unsigned char>(min(max(gradient_magnitude, 0.0f), 255.0f));

        unsigned char gray_value = static_cast<unsigned char>(result[pixel_index] * 0.3f + result[pixel_index + 1] * 0.59f + result[pixel_index + 2] * 0.11f);
        result[pixel_index + 2] = min(255, gray_value + 40);
        result[pixel_index + 1] = min(255, gray_value + 20);
        result[pixel_index] = min(255, gray_value);
    }
}

extern "C" __global__ void color_detection_kernel(int width, int height, unsigned char* pixels, unsigned char* result, unsigned char* color) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height) {
		int pixel_index = idx * 3;

		bool color_detected = 
            (pixels[pixel_index + 2] <= color[0] + 20 && pixels[pixel_index + 2] >= color[0] - 20) &&
			(pixels[pixel_index + 1] <= color[1] + 20 && pixels[pixel_index + 1] >= color[1] - 20) &&
			(pixels[pixel_index] <= color[2] + 20 && pixels[pixel_index] >= color[2] - 20);

		result[pixel_index + 2] = color_detected ? color[0] : 128;
		result[pixel_index + 1] = color_detected ? color[1] : 128;
		result[pixel_index] = color_detected ? color[2] : 128;
    }
}





extern "C" __declspec(dllexport) void GenerateHistograms(int width, int height, unsigned char* pixels, int* red_histogram, int* green_histogram, int* blue_histogram) {
    unsigned char* d_pixels;
    int* d_red_histogram;
    int* d_green_histogram;
    int* d_blue_histogram;

    hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
    hipMalloc((void**)&d_red_histogram, 256 * sizeof(int));
    hipMalloc((void**)&d_green_histogram, 256 * sizeof(int));
    hipMalloc((void**)&d_blue_histogram, 256 * sizeof(int));

    hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemset(d_red_histogram, 0, 256 * sizeof(int));
    hipMemset(d_green_histogram, 0, 256 * sizeof(int));
    hipMemset(d_blue_histogram, 0, 256 * sizeof(int));

    int blockSize = 256;
    int numBlocks = (width * height + blockSize - 1) / blockSize;
    generate_histogram_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_red_histogram, d_green_histogram, d_blue_histogram);

    hipMemcpy(red_histogram, d_red_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(green_histogram, d_green_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(blue_histogram, d_blue_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_pixels);
    hipFree(d_red_histogram);
    hipFree(d_green_histogram);
    hipFree(d_blue_histogram);
}

extern "C" __declspec(dllexport) void GrayscaleFilter(int width, int height, unsigned char* pixels, unsigned char* result) {
    unsigned char* d_pixels;
	unsigned char* d_result;

    hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
	grayscale_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void NegativeFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
	negative_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void SepiaFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
	sepia_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void GaussianFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
	gaussian_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void EmbossFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
	emboss_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void EdgeFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    edge_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void ThermalFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    thermal_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void ContrastFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    contrast_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void SharpenFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    sharpen_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void NoiseFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    noise_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void TiltFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    tilt_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void SketchFilter(int width, int height, unsigned char* pixels, unsigned char* result) {

    unsigned char* d_pixels;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    sketch_filter_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_result);
}

extern "C" __declspec(dllexport) void ColorDetection(int width, int height, unsigned char* pixels, unsigned char* result, unsigned char* color) {

    unsigned char* d_pixels;
    unsigned char* d_color;
	unsigned char* d_result;

	hipMalloc((void**)&d_pixels, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_result, width * height * 3 * sizeof(unsigned char));
	hipMalloc((void**)&d_color, 3 * sizeof(unsigned char));

	hipMemcpy(d_pixels, pixels, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_color, color, 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (width * height + blockSize - 1) / blockSize;
    color_detection_kernel << <numBlocks, blockSize >> > (width, height, d_pixels, d_result, d_color);

	hipMemcpy(result, d_result, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_color);
	hipFree(d_result);
}